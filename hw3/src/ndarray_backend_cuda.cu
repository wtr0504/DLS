#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

#include <iostream>
#include <sstream>
#include <math.h> 
#include <limits>
#include <float.h>

namespace needle {
namespace cuda {

#define BASE_THREAD_NUM 256

#define TILE 4
typedef float scalar_t;
const size_t ELEM_SIZE = sizeof(scalar_t);

struct CudaArray {
  CudaArray(const size_t size) {
    hipError_t err = hipMalloc(&ptr, size * ELEM_SIZE);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
    this->size = size;
  }
  ~CudaArray() { hipFree(ptr); }
  size_t ptr_as_int() { return (size_t)ptr; }
  
  scalar_t* ptr;
  size_t size;
};

struct CudaDims {
  dim3 block, grid;
};

CudaDims CudaOneDim(size_t size) {
  /**
   * Utility function to get cuda dimensions for 1D call
   */
  CudaDims dim;
  size_t num_blocks = (size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}

CudaDims CudaNDims(size_t size, size_t perBlockOpNums){
  CudaDims dim;
  size_t num_blocks = ((size + BASE_THREAD_NUM - 1) / BASE_THREAD_NUM  + perBlockOpNums - 1)/ perBlockOpNums;
  dim.block = dim3(BASE_THREAD_NUM, 1, 1);
  dim.grid = dim3(num_blocks, 1, 1);
  return dim;
}


#define MAX_VEC_SIZE 8
struct CudaVec {
  uint32_t size;
  int32_t data[MAX_VEC_SIZE];
};

CudaVec VecToCuda(const std::vector<int32_t>& x) {
  CudaVec shape;
  if (x.size() > MAX_VEC_SIZE) throw std::runtime_error("Exceeded CUDA supported max dimesions");
  shape.size = x.size();
  for (size_t i = 0; i < x.size(); i++) {
    shape.data[i] = x[i];
  }
  return shape;
}

////////////////////////////////////////////////////////////////////////////////
// Fill call
////////////////////////////////////////////////////////////////////////////////

__global__ void FillKernel(scalar_t* out, scalar_t val, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = val;
}

void Fill(CudaArray* out, scalar_t val) {
  CudaDims dim = CudaOneDim(out->size);
  FillKernel<<<dim.grid, dim.block>>>(out->ptr, val, out->size);
}

////////////////////////////////////////////////////////////////////////////////
// Compact and setitem cals
////////////////////////////////////////////////////////////////////////////////

// Untility function to convert contiguous index i to memory location from strides

__device__ size_t index_2_location(size_t index,CudaVec shape,
                              CudaVec strides, size_t offset){
  size_t idx[MAX_VEC_SIZE];
  size_t pre_size = 1,cur_size = 1;
  size_t pos = offset;
  size_t ndim = shape.size;
  for(int i = ndim - 1;i >= 0;i--){
    cur_size = pre_size * shape.data[i];
    idx[i] = index % cur_size / pre_size;
    pre_size = cur_size;
  }
  for(int i = 0;i < ndim;i++){
    pos += idx[i] * strides.data[i];
  }
  return pos;
}


__global__ void CompactKernel(const scalar_t* a, scalar_t* out, size_t size, CudaVec shape,
                              CudaVec strides, size_t offset) {
  /**
   * The CUDA kernel for the compact opeation.  This should effectively map a single entry in the 
   * non-compact input a, to the corresponding item (at location gid) in the compact array out.
   * 
   * Args:
   *   a: CUDA pointer to a array
   *   out: CUDA point to out array
   *   size: size of out array
   *   shape: vector of shapes of a and out arrays (of type CudaVec, for past passing to CUDA kernel)
   *   strides: vector of strides of out array
   *   offset: offset of out array
   */
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;

  /// BEGIN SOLUTION
  if(gid < size){
    out[gid] = a[index_2_location(gid,shape,strides,offset)];
  }
  /// END SOLUTION
}

void Compact(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
             std::vector<int32_t> strides, size_t offset) {
  /**
   * Compact an array in memory.  Unlike the C++ version, in CUDA this will primarily call the 
   * relevant CUDA kernel.  In this case, we illustrate how you should set this up (i.e., we give 
   * you the code for this fuction, and also the prototype for the CompactKernel() function).  For
   * the functions after this, however, you'll need to define these kernels as you see fit to 
   * execute the underlying function.
   * 
   * Args:
   *   a: non-compact represntation of the array, given as input
   *   out: compact version of the array to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *a* array (not out, which has compact strides)
   *   offset: offset of the *a* array (not out, which has zero offset, being compact)
   */

  // Nothing needs to be added here
  CudaDims dim = CudaOneDim(out->size);
  CompactKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size, VecToCuda(shape),
                                         VecToCuda(strides), offset);
}

__global__ void EwiseSetitemKernel(const scalar_t *a,scalar_t *out,size_t size, CudaVec shape,
                              CudaVec strides, size_t offset){
  size_t gid = blockDim.x * blockIdx.x + threadIdx.x;

  if(gid < size){
    out[index_2_location(gid,shape,strides,offset)] = a[gid];
  }
}


void EwiseSetitem(const CudaArray& a, CudaArray* out, std::vector<int32_t> shape,
                  std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items in a (non-compact) array using CUDA.  Yyou will most likely want to implement a
   * EwiseSetitemKernel() function, similar to those above, that will do the actual work.
   * 
   * Args:
   *   a: _compact_ array whose items will be written to out
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension for a and out
   *   strides: strides of the *out* array (not a, which has compact strides)
   *   offset: offset of the *out* array (not a, which has zero offset, being compact)
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(a.size);
  EwiseSetitemKernel<<<dim.grid,dim.block>>>(a.ptr,out->ptr,a.size,VecToCuda(shape),VecToCuda(strides),offset);

  
  /// END SOLUTION
}

__global__ void ScalarSetitemKernel(scalar_t *out,const scalar_t val,size_t size, CudaVec shape,
                              CudaVec strides, size_t offset){
  size_t gid = blockDim.x * blockIdx.x + threadIdx.x;

  if(gid < size){
    out[index_2_location(gid,shape,strides,offset)] = val;
  }
}

void ScalarSetitem(size_t size, scalar_t val, CudaArray* out, std::vector<int32_t> shape,
                   std::vector<int32_t> strides, size_t offset) {
  /**
   * Set items is a (non-compact) array
   * 
   * Args:
   *   size: number of elements to write in out array (note that this will note be the same as
   *         out.size, because out is a non-compact subset array);  it _will_ be the same as the 
   *         product of items in shape, but covenient to just pass it here.
   *   val: scalar value to write to
   *   out: non-compact array whose items are to be written
   *   shape: shapes of each dimension of out
   *   strides: strides of the out array
   *   offset: offset of the out array
   */
  /// BEGIN SOLUTION
  CudaDims dim = CudaOneDim(out->size);
  ScalarSetitemKernel<<<dim.grid,dim.block>>>(out->ptr,val,out->size,VecToCuda(shape),VecToCuda(strides),offset);

  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////


__global__ void EwiseAddKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + b[gid];
}

void EwiseAdd(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA arrays.
   * Args:
   *   a: Input array 'a' to be added
   *   b: Input array 'b' to be added
   *   out: Output array to store the result of 'a + b'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Kernel will execute on 'dim.grid' blocks, each containing 'dim.block' threads.
  EwiseAddKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarAddKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] + val;
}

void ScalarAdd(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add a scalar value to every element of a CUDA array.
   * Args:
   *   a: Input array 'a'
   *   val: Scalar value to be added
   *   out: Output array to store the result of 'a + val'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Launch the ScalarAddKernel that will add the scalar 'val' to each element of array 'a', 
  // and store the result in array 'out'.
  ScalarAddKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

/**
 * In the code the follows, use the above template to create analogous elementise
 * and and scalar operators for the following functions.  See the numpy backend for
 * examples of how they should work.
 *   - EwiseMul, ScalarMul
 *   - EwiseDiv, ScalarDiv
 *   - ScalarPower
 *   - EwiseMaximum, ScalarMaximum
 *   - EwiseEq, ScalarEq
 *   - EwiseGe, ScalarGe
 *   - EwiseLog
 *   - EwiseExp
 *   - EwiseTanh
 *
 * If you implement all these naively, there will be a lot of repeated code, so
 * you are welcome (but not required), to use macros or templates to define these
 * functions (however you want to do so, as long as the functions match the proper)
 * signatures above.
 */

__global__ void EwiseMulKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * b[gid];
}

void EwiseMul(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA arrays.
   * Args:
   *   a: Input array 'a' to be added
   *   b: Input array 'b' to be added
   *   out: Output array to store the result of 'a + b'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Kernel will execute on 'dim.grid' blocks, each containing 'dim.block' threads.
  EwiseMulKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMulKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] * val;
}

void ScalarMul(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add a scalar value to every element of a CUDA array.
   * Args:
   *   a: Input array 'a'
   *   val: Scalar value to be added
   *   out: Output array to store the result of 'a + val'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Launch the ScalarAddKernel that will add the scalar 'val' to each element of array 'a', 
  // and store the result in array 'out'.
  ScalarMulKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}

__global__ void EwiseDivKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / b[gid];
}

void EwiseDiv(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA arrays.
   * Args:
   *   a: Input array 'a' to be added
   *   b: Input array 'b' to be added
   *   out: Output array to store the result of 'a + b'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Kernel will execute on 'dim.grid' blocks, each containing 'dim.block' threads.
  EwiseDivKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarDivKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = a[gid] / val;
}

void ScalarDiv(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add a scalar value to every element of a CUDA array.
   * Args:
   *   a: Input array 'a'
   *   val: Scalar value to be added
   *   out: Output array to store the result of 'a + val'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Launch the ScalarAddKernel that will add the scalar 'val' to each element of array 'a', 
  // and store the result in array 'out'.
  ScalarDivKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void ScalarPowerKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = powf(a[gid],val);
}

void ScalarPower(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add a scalar value to every element of a CUDA array.
   * Args:
   *   a: Input array 'a'
   *   val: Scalar value to be added
   *   out: Output array to store the result of 'a + val'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Launch the ScalarAddKernel that will add the scalar 'val' to each element of array 'a', 
  // and store the result in array 'out'.
  ScalarPowerKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void EwiseMaximumKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = max(a[gid],b[gid]);
}

void EwiseMaximum(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  /**
   * Add together two CUDA arrays.
   * Args:
   *   a: Input array 'a' to be added
   *   b: Input array 'b' to be added
   *   out: Output array to store the result of 'a + b'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Kernel will execute on 'dim.grid' blocks, each containing 'dim.block' threads.
  EwiseMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarMaximumKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  // Calculate the global index of the thread.
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = max(a[gid],val);
}

void ScalarMaximum(const CudaArray& a, scalar_t val, CudaArray* out) {
  /**
   * Add a scalar value to every element of a CUDA array.
   * Args:
   *   a: Input array 'a'
   *   val: Scalar value to be added
   *   out: Output array to store the result of 'a + val'
   */
  CudaDims dim = CudaOneDim(out->size);

  // Launch the ScalarAddKernel that will add the scalar 'val' to each element of array 'a', 
  // and store the result in array 'out'.
  ScalarMaximumKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}


__global__ void EwiseEqKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = static_cast<scalar_t> (a[gid] == b[gid]);
}

void EwiseEq(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);

  EwiseEqKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarEqKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = static_cast<scalar_t> (a[gid] == val);
}

void ScalarEq(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarEqKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}



__global__ void EwiseGeKernel(const scalar_t* a, const scalar_t* b, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = static_cast<scalar_t> (a[gid] >= b[gid]);
}

void EwiseGe(const CudaArray& a, const CudaArray& b, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);

  EwiseGeKernel<<<dim.grid, dim.block>>>(a.ptr, b.ptr, out->ptr, out->size);
}

__global__ void ScalarGeKernel(const scalar_t* a, scalar_t val, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = static_cast<scalar_t> (a[gid] >= val);
}

void ScalarGe(const CudaArray& a, scalar_t val, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);
  ScalarGeKernel<<<dim.grid, dim.block>>>(a.ptr, val, out->ptr, out->size);
}



__global__ void EwiseLogKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = logf(a[gid]);
}

void EwiseLog(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);

  EwiseLogKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}

__global__ void EwiseExpKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = expf(a[gid]);
}

void EwiseExp(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);

  EwiseExpKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}


__global__ void EwiseTanhKernel(const scalar_t* a, scalar_t* out, size_t size) {
  size_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < size) out[gid] = tanhf(a[gid]);
}

void EwiseTanh(const CudaArray& a, CudaArray* out) {
  CudaDims dim = CudaOneDim(out->size);

  EwiseTanhKernel<<<dim.grid, dim.block>>>(a.ptr, out->ptr, out->size);
}



////////////////////////////////////////////////////////////////////////////////
// Elementwise and scalar operations
////////////////////////////////////////////////////////////////////////////////

__global__ void MatmulKernel(const scalar_t *a,const scalar_t *b,scalar_t *out,size_t M,size_t N,size_t P){

  constexpr int BM = 32;
  constexpr int BN = 32;
  constexpr int BP = 32;

  __shared__ scalar_t matA[BM][BN];
  __shared__ scalar_t matB[BN][BP];

  const int tidr = threadIdx.x;
  const int tidc = threadIdx.y;
  const int bidr = blockIdx.x * blockDim.x;
  const int bidc = blockIdx.y * blockDim.y;

  for(int j = 0; j < M;j += BN){
    if(tidr + bidr < BM && j + tidc < BN){
      matA[tidr][tidc] = a[(tidr + bidr) * BN + j + tidc];
    } else {
      matA[tidr][tidc] = 0.0;
    }

    if(tidr + j < BN && tidc + bidc < BP){
      matB[tidr][tidc] = b[(tidr + j) * BP + tidc + bidc];
    } else {
      matB[tidr][tidc] = 0.0;
    }
    __syncthreads();
    float res = 0.0;
    for(int i = 0; i < 32;i++){
      res += matA[tidr][i] * matB[i][tidc];
    }
    __syncthreads();
    if(tidc + bidc < P && tidr + bidr < M){
      out[(tidr + bidr) * P + tidc + bidc] += res;
    }
  }


}

void Matmul(const CudaArray& a, const CudaArray& b, CudaArray* out, uint32_t M, uint32_t N,
            uint32_t P) {
  /**
   * Multiply two (compact) matrices into an output (also comapct) matrix.  You will want to look
   * at the lecture and notes on GPU-based linear algebra to see how to do this.  Since ultimately
   * mugrade is just evaluating correctness, you _can_ implement a version that simply parallelizes
   * over (i,j) entries in the output array.  However, to really get the full benefit of this
   * problem, we would encourage you to use cooperative fetching, shared memory register tiling, 
   * and other ideas covered in the class notes.  Note that unlike the tiled matmul function in
   * the CPU backend, here you should implement a single function that works across all size
   * matrices, whether or not they are a multiple of a tile size.  As with previous CUDA
   * implementations, this function here will largely just set up the kernel call, and you should
   * implement the logic in a separate MatmulKernel() call.
   * 
   *
   * Args:
   *   a: compact 2D array of size m x n
   *   b: comapct 2D array of size n x p
   *   out: compact 2D array of size m x p to write the output to
   *   M: rows of a / out
   *   N: columns of a / rows of b
   *   P: columns of b / out
   */

  /// BEGIN SOLUTION
  dim3 blocks((M + 32 - 1)/32,(P + 32-1)/32);
  dim3 threadsPerblock(32,32);
  MatmulKernel<<<blocks,threadsPerblock>>>(a.ptr,b.ptr,out->ptr,M,N,P);
  
  /// END SOLUTION
}

////////////////////////////////////////////////////////////////////////////////
// Max and sum reductions
////////////////////////////////////////////////////////////////////////////////


// __global__ void ReduceMaxKernel(const scalar_t* a, scalar_t *out,size_t n){
//   extern __shared__ scalar_t block_data[];
//   size_t idx = blockDim.x * blockIdx.x * 4 + threadIdx.x;
//   size_t tid = threadIdx.x;
//   if(tid >= n) return;
//   scalar_t *input = a + blockDim.x * blockIdx.x * 4;

//   if(idx + 3 * blockDim.x < n){
//     scalar a1 = a[idx];
//     a1 = max(a1,a[idx + blockDim.x]);
//     a1 = max(a1,a[idx + 2 * blockDim.x]);
//     a1 = max(a1,a[idx + 3 * blockDim.x]);

//     block_data[tid] = a1;
//   }
//   __syncthreads();

// }

__global__ void ReduceMaxKernel(const scalar_t* a, scalar_t *out,size_t n,size_t op_times){
  extern __shared__ scalar_t block_data[];
  // size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  size_t tid = threadIdx.x;

  scalar_t tmp = (tid < n) ? a[tid] : -FLT_MAX ;
  for(int i = tid + blockDim.x;i < n;i += blockDim.x){
    tmp= max(a[i],tmp);
  }
  block_data[tid] = tmp;

  __syncthreads();

  if(n > blockDim.x / 2)
    for(size_t strides = blockDim.x / 2; strides > 32; strides >>= 1){
      if(tid < strides){
        block_data[tid] = max(block_data[tid],block_data[tid + strides]);
      }
      __syncthreads();
    }

  if(tid < 32){
    volatile scalar_t* vmem = block_data;
    vmem[tid] = max(vmem[tid+32],vmem[tid]);
		vmem[tid] = max(vmem[tid+16],vmem[tid]);
		vmem[tid] = max(vmem[tid+8],vmem[tid]);
		vmem[tid] = max(vmem[tid+4],vmem[tid]);
		vmem[tid] = max(vmem[tid+2],vmem[tid]);
		vmem[tid] = max(vmem[tid+1],vmem[tid]);
  }
  if (tid == 0)
		out[0] = block_data[0];
}


void ReduceMax(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking maximum over `reduce_size` contiguous blocks.  Even though it is inefficient,
   * for simplicity you can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
  size_t op_times = (reduce_size + BASE_THREAD_NUM - 1)/ BASE_THREAD_NUM;
  CudaDims dim = {BASE_THREAD_NUM,reduce_size};
  const int n_streams = 4;
  hipStream_t streams[n_streams];
  for (int s = 0; s < n_streams; s++) {
      hipStreamCreate(&streams[s]);
  }

  size_t task_id = 0;
  for(size_t i = 0; i < out->size; i++) {
      int stream_id = task_id % n_streams;
      ReduceMaxKernel<<<1,
                        dim.block,
                        BASE_THREAD_NUM * sizeof(scalar_t),
                        streams[stream_id]>>>(a.ptr + i * reduce_size, out->ptr + i, reduce_size, op_times);
      task_id++;
  }

  for (int s = 0; s < n_streams; s++) {
      hipStreamSynchronize(streams[s]);
      hipStreamDestroy(streams[s]);
  }
  /// END SOLUTION
}


__global__ void ReduceSumKernel(const scalar_t* a, scalar_t *out,size_t n,size_t op_times){
  extern __shared__ scalar_t block_data[];
  // size_t idx = blockDim.x * blockIdx.x + threadIdx.x;
  size_t tid = threadIdx.x;

  scalar_t tmp = 0.0;
  for(int i = tid;i < n;i += blockDim.x){
    tmp += a[i];
  }
  block_data[tid] = tmp;

  __syncthreads();

  if(n > blockDim.x / 2)
    for(size_t strides = blockDim.x / 2; strides > 32; strides >>= 1){
      if(tid < strides){
        block_data[tid] += block_data[tid + strides];
      }
      __syncthreads();
    }

  if(tid < 32){
    volatile scalar_t* vmem = block_data;
		vmem[tid]+=vmem[tid+32];
		vmem[tid]+=vmem[tid+16];
		vmem[tid]+=vmem[tid+8];
		vmem[tid]+=vmem[tid+4];
		vmem[tid]+=vmem[tid+2];
		vmem[tid]+=vmem[tid+1];
  }
  if (tid == 0)
		out[0] = block_data[0];
}


void ReduceSum(const CudaArray& a, CudaArray* out, size_t reduce_size) {
  /**
   * Reduce by taking summation over `reduce_size` contiguous blocks.  Again, for simplicity you 
   * can perform each reduction in a single CUDA thread.
   * 
   * Args:
   *   a: compact array of size a.size = out.size * reduce_size to reduce over
   *   out: compact array to write into
   *   redice_size: size of the dimension to reduce over
   */
  /// BEGIN SOLUTION
    size_t op_times = (reduce_size + BASE_THREAD_NUM - 1)/ BASE_THREAD_NUM;
  CudaDims dim = {BASE_THREAD_NUM,reduce_size};
  const int n_streams = 4;
  hipStream_t streams[n_streams];
  for (int s = 0; s < n_streams; s++) {
      hipStreamCreate(&streams[s]);
  }

  size_t task_id = 0;
  for(size_t i = 0; i < out->size; i++) {
      int stream_id = task_id % n_streams;
      ReduceSumKernel<<<1,
                        dim.block,
                        BASE_THREAD_NUM * sizeof(scalar_t),
                        streams[stream_id]>>>(a.ptr + i * reduce_size, out->ptr + i, reduce_size, op_times);
      task_id++;
  }

  for (int s = 0; s < n_streams; s++) {
      hipStreamSynchronize(streams[s]);
      hipStreamDestroy(streams[s]);
  }
  /// END SOLUTION
}


}  // namespace cuda
}  // namespace needle

PYBIND11_MODULE(ndarray_backend_cuda, m) {
  namespace py = pybind11;
  using namespace needle;
  using namespace cuda;

  m.attr("__device_name__") = "cuda";
  m.attr("__tile_size__") = TILE;

  py::class_<CudaArray>(m, "Array")
      .def(py::init<size_t>(), py::return_value_policy::take_ownership)
      .def_readonly("size", &CudaArray::size)
      .def("ptr", &CudaArray::ptr_as_int);

  // return numpy array, copying from CPU
  m.def("to_numpy", [](const CudaArray& a, std::vector<size_t> shape, std::vector<size_t> strides,
                       size_t offset) {
    std::vector<size_t> numpy_strides = strides;
    std::transform(numpy_strides.begin(), numpy_strides.end(), numpy_strides.begin(),
                   [](size_t& c) { return c * ELEM_SIZE; });

    // copy memory to host
    scalar_t* host_ptr = (scalar_t*)std::malloc(a.size * ELEM_SIZE);
    if (host_ptr == 0) throw std::bad_alloc();
    hipError_t err = hipMemcpy(host_ptr, a.ptr, a.size * ELEM_SIZE, hipMemcpyDeviceToHost);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));

    // return numpy array
    py::capsule deallocate_buffer(host_ptr, [](void* p) { free(p); });
    return py::array_t<scalar_t>(shape, numpy_strides, host_ptr + offset, deallocate_buffer);
  });

  // copy numpy array to GPU
  m.def("from_numpy", [](py::array_t<scalar_t> a, CudaArray* out) {
    hipError_t err =
        hipMemcpy(out->ptr, a.request().ptr, out->size * ELEM_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess) throw std::runtime_error(hipGetErrorString(err));
  });

  m.def("fill", Fill);
  m.def("compact", Compact);
  m.def("ewise_setitem", EwiseSetitem);
  m.def("scalar_setitem", ScalarSetitem);
  m.def("ewise_add", EwiseAdd);
  m.def("scalar_add", ScalarAdd);

  m.def("ewise_mul", EwiseMul);
  m.def("scalar_mul", ScalarMul);
  m.def("ewise_div", EwiseDiv);
  m.def("scalar_div", ScalarDiv);
  m.def("scalar_power", ScalarPower);

  m.def("ewise_maximum", EwiseMaximum);
  m.def("scalar_maximum", ScalarMaximum);
  m.def("ewise_eq", EwiseEq);
  m.def("scalar_eq", ScalarEq);
  m.def("ewise_ge", EwiseGe);
  m.def("scalar_ge", ScalarGe);

  m.def("ewise_log", EwiseLog);
  m.def("ewise_exp", EwiseExp);
  m.def("ewise_tanh", EwiseTanh);

  m.def("matmul", Matmul);

  m.def("reduce_max", ReduceMax);
  m.def("reduce_sum", ReduceSum);
}
